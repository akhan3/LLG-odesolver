#include "hip/hip_runtime.h"
/* ==========================================================================
 * kernel.cu
 *
 * This is the CUDA source file that is compiled separately by MATLAB routine
 * and then linked to by MEX.
 * User must edit by this file to model different problems.
 *
 * Project:     Massive ODE Solver
 * Author:      Aamir Ahmed Khan (akhan3@nd.edu)
 * Copyright:   2012, University of Notre Dame
 * =========================================================================*/

#include "globals.h"
#include <cutil_inline.h>


/* Device constant memory */
__device__ __constant__ single *bcMtop_c, *bcMbot_c, *bcMrig_c, *bcMlef_c;


/* IMPORTANT: Don't mess with this function!!! Just use it */
/*! Picks the correct neighbors based on the location
 *      Picks the neighbor if exists otherwise pick the corresponding boundary condition */
__device__
void pickNeighbors_d( single **S_top, single **S_bot, single **S_rig, single **S_lef,
                      single *bcMtop, single *bcMbot, single *bcMrig, single *bcMlef,
                      single *S, const int ix, const int iy,
                      const int Nx, const int Ny, const int Ns )
{
    /* Here, top means maximum iy coordinate, not the first matrix row
     *       This code follows xy-coordinate axes convention */
    // if at top-most row, use top boundary condition
    *S_top = (iy == Ny-1)  ?  &bcMtop[ix*Ns]  :  &S[(iy+1)*Ns+ ix   *Ny*Ns]; // +y (wrt to xy-cood axes)
    // if at bottom-most row, use bottom boundary condition
    *S_bot = (iy == 0)        ?  &bcMbot[ix*Ns]  :  &S[(iy-1)*Ns+ ix   *Ny*Ns]; // -y (wrt to xy-cood axes)
    // if at right-most column, use right boundary condition
    *S_rig = (ix == Nx-1)  ?  &bcMrig[iy*Ns]  :  &S[ iy   *Ns+(ix+1)*Ny*Ns]; // +x
    // if at left-most column, use left boundary condition
    *S_lef = (ix == 0)        ?  &bcMlef[iy*Ns]  :  &S[ iy   *Ns+(ix-1)*Ny*Ns]; // -x
}


/*! Kernel definition */
__global__ void
kernel( single *Sprime, single *S,
        single *bcMtop, single *bcMbot, single *bcMrig, single *bcMlef,
        const int Nx, const int Ny, const int Ns,
        const single dx, const single dy )
{
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if(n >= Ny * Nx)
        return;

    // extract ix and iy indices from the single index n
    int iy = n % Ny;
    int ix = (n-iy) / Ny;

    /* Pick neighbours */
    single *S_top, *S_bot, *S_rig, *S_lef;  // pointers to neighbors
    // this function must be called to take care of boundary conditions and pick proper neighbors
    pickNeighbors_d(&S_top, &S_bot, &S_rig, &S_lef, bcMtop, bcMbot, bcMrig, bcMlef, S, ix, iy, Nx, Ny, Ns);


    /* Execute the differential equations */
    // Variable order : S[0] - Hx        S[1] - Hy       S[2] - Ez
    // Calculate Laplacian
    single *S_cen = &S[n * Ns];   // pointer to variable
    single DHx = -1.0f*(S_top[2] - 2.0f*S_cen[2] + S_bot[2]) / (dy*dy);
    single DHy = +1.0f*(S_lef[2] - 2.0f*S_cen[2] + S_rig[2]) / (dx*dx);
    single DEz = -1.0f*( (S_lef[1] - 2.0f*S_cen[1] + S_rig[1]) / (dx*dx) - (S_top[0] - 2.0f*S_cen[0] + S_bot[0]) / (dy*dy) );

    /* Assign the derivatives */
    Sprime[n*Ns+0] = DHx;
    Sprime[n*Ns+1] = DHy;
    Sprime[n*Ns+2] = DEz;
}


/*! Evaluates the slopes from the vector differential equation.
 *      This is the major function which the user has to modify to model their
 *      own problem.
 *  \param Sprime Return pointer for computed dS/dt vector field
 *  \param sp Simulation parameters
 *  \param S State vector */
void differentiateStateVectorField_d( single *Sprime, single *S, simParam sp )
{
    int Nxy = sp.Ny * sp.Nx;    // size of array

    /* set up and allocate device memory */
    cutilSafeCall( hipSetDevice(sp.useGPUnum) );
    single *Sprime_d = NULL;
    single *S_d = NULL;
    single *bcMtop_d = NULL;
    single *bcMbot_d = NULL;
    single *bcMrig_d = NULL;
    single *bcMlef_d = NULL;
    cutilSafeCall( hipMalloc( (void**)&Sprime_d,   sp.Ns*Nxy*sizeof(single) ) );
    cutilSafeCall( hipMalloc( (void**)&S_d,        sp.Ns*Nxy*sizeof(single) ) );
    cutilSafeCall( hipMalloc( (void**)&bcMtop_d,   sp.Ns*sp.Nx*sizeof(single) ) );
    cutilSafeCall( hipMalloc( (void**)&bcMbot_d,   sp.Ns*sp.Nx*sizeof(single) ) );
    cutilSafeCall( hipMalloc( (void**)&bcMrig_d,   sp.Ns*sp.Ny*sizeof(single) ) );
    cutilSafeCall( hipMalloc( (void**)&bcMlef_d,   sp.Ns*sp.Ny*sizeof(single) ) );
    assert(Sprime_d != NULL && S_d != NULL && bcMtop_d != NULL && bcMbot_d != NULL && bcMrig_d != NULL && bcMlef_d != NULL);

    /* copy State Variables to device global memory */
    cutilSafeCall( hipMemset( Sprime_d, 0, sp.Ns*Nxy*sizeof(single) ) );
    cutilSafeCall( hipMemcpy( S_d, S, sp.Ns*Nxy*sizeof(single), hipMemcpyHostToDevice ) );
    cutilSafeCall( hipMemcpy( bcMtop_d, sp.bcMtop, sp.Ns*sp.Nx*sizeof(single), hipMemcpyHostToDevice ) );
    cutilSafeCall( hipMemcpy( bcMbot_d, sp.bcMbot, sp.Ns*sp.Nx*sizeof(single), hipMemcpyHostToDevice ) );
    cutilSafeCall( hipMemcpy( bcMrig_d, sp.bcMrig, sp.Ns*sp.Ny*sizeof(single), hipMemcpyHostToDevice ) );
    cutilSafeCall( hipMemcpy( bcMlef_d, sp.bcMlef, sp.Ns*sp.Ny*sizeof(single), hipMemcpyHostToDevice ) );

    /* set up kernel parameters */
    #ifdef __DEVICE_EMULATION__
        #define DIM 64
    #else
        #define DIM 512
    #endif
    dim3 grid = ceil(Nxy / (single)DIM);
    dim3 threads(DIM, 1, 1);
    assert(threads.x <= DIM);    // max_threads_per_block

    /* launch the kernel */
    kernel <<<grid, threads>>> (Sprime_d, S_d, bcMtop_d, bcMbot_d, bcMrig_d, bcMlef_d, sp.Nx, sp.Ny, sp.Ns, sp.dx, sp.dy );

    /* copy Sprime_d (result of the kernel) back to host main memory */
    cutilSafeCall( hipMemcpy( Sprime, Sprime_d, sp.Ns*Nxy*sizeof(single), hipMemcpyDeviceToHost ) );

    /* clean-up */
    // must deallocate all the reserved memory, otherwise huge performance penalty!!!
    cutilSafeCall( hipFree(Sprime_d) );
    cutilSafeCall( hipFree(S_d) );
    cutilSafeCall( hipFree(bcMtop_d) );
    cutilSafeCall( hipFree(bcMbot_d) );
    cutilSafeCall( hipFree(bcMrig_d) );
    cutilSafeCall( hipFree(bcMlef_d) );
}
